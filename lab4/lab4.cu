#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cmath>
#include <ctime> 
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

using namespace std;

__global__
void countnewmatrix(double* mas, double* anew, size_t n) //считаем новую матрицу
{
    size_t i = blockIdx.x; //получаем индексы, блок
    size_t j = threadIdx.x; //нить

    if (!(blockIdx.x == 0 || threadIdx.x == 0) && (blockIdx.x >= 0 && blockIdx.x < n && threadIdx.x >= 0 && threadIdx.x < n))
        anew[i * n + j] = (mas[i * n + j - 1] + mas[(i - 1) * n + j] + mas[(i + 1) * n + j] + mas[i * n + j + 1]) * 0.25; //считаем поэлементно

}

__global__
void finderr(double* mas, double* anew, double* outMatrix, size_t n) //обновляем значение ошибки
{				//разм-ть блока
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x; //получаем индекс элемента

    if (!(blockIdx.x == 0 || threadIdx.x == 0) && (blockIdx.x >= 0 && blockIdx.x < size && threadIdx.x >= 0 && threadIdx.x < size))
        outMatrix[idx] = fabs(anew[idx] - mas[idx]); //берём по модулю

}

int main(int arg, char** argv) {
    int N = atoi(argv[1]); //параметр, размер сетки
    int ITER = std::atoi(argv[2]); //параметр, макс количество итераций
    float ACC = std::atof(argv[3]); //параметр, мин значение ошибки
    //защита от пользователя
    if (N == 0 || N < 0){
	std::cout<<"N error"<<std::endl;
	return EXIT_FAILURE;
    }

    if (ITER > 1000000 || ITER < 0)
    {
	std::cout<<"ITER error"<<std::endl;
	return EXIT_FAILURE;
    }

    if (ACC < 0.0000001){
	std::cout<<"ACC error"<<std::endl;
	return EXIT_FAILURE;
    }


    double* mas = new double[N * N]; //создаем массив, который будет содержать старую копию  
    double* anew = new double[N * N]; //массив, который будет содержать новую версию

    for (int i = 0; i < N * N; i++)  //заполняю оба массива нулями, чтобы избавиться от возможного мусора
    {
        mas[i] = 0;
        anew[i] = 0;
    }

    int rep = 0; //инициализация переменной, отвечающей за отслеживания количества итераций 
    double err = 1.0; //инициализация переменной, отвечающей за отслеживания ошибки на определённой итерации

    mas[0] = 10; //левый верхний угол матрицы
    mas[N - 1] = 20; //правый верхний угол матрицы
    mas[N * (N - 1)] = 20; //нижний левый угол матрицы
    mas[N * N-1] = 30; //нижний правый угол матрицы

    clock_t befin = clock(); //начнаю отсчёт времени инициализации 
    for (int i = 1; i < N-1; i++) //запоняю верхнюю границу матрицы
        mas[i] = mas[i-1] + (mas[N - 1] - mas[0]) / (N-1);

    for (int i = 1; i < N-1; i++) //заполняю левую, правую и нижнуюю границы матрицы
    {
        mas[N * (N - 1) + i] = mas[N * (N - 1) + i-1] + (mas[N * N-1] - mas[N * (N - 1)]) / (N-1); 
        mas[N*i] = mas[N*i-N] + (mas[N * N-1] - mas[N-1]) / (N-1);
        mas[(N)*i+(N-1)] = mas[(N)*(i-1)+(N-1)] + (mas[N * N - 1] - mas[N - 1]) / (N-1);
    }
  
	for (int i = 0; i < N*N; i++) //копирую
		anew[i] = mas[i];

        cout << "Initialization Time: " << 1.0 * (clock() - befin) / CLOCKS_PER_SEC << endl; //вывод потраченного на инициализацию времени
        clock_t befca = clock();//начало отсчёта для выполнения вычислений

	//создаю переменные и матрицы для работы на девайсе
        double* mas_dev, * anew_dev, * deviceError, * errorMatrix, * tempStorage = NULL;
        size_t tempStorageSize = 0;
	
	//выделяю память для матриц и переменных
        hipMalloc((void**)(&mas_dev), sizeof(double) * N * N);
        hipMalloc((void**)(&anew_dev), sizeof(double) * N * N);
        hipMalloc((void**)&deviceError, sizeof(double));
        hipMalloc((void**)&errorMatrix, sizeof(double) * N * N);
	
	//выделние памяти для временного хранения CUB
 	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, N * N); //NULL
        hipMalloc(&tempStorage, tempStorageSize);
	
	//копирую из хоста на девайс
        hipMemcpy(mas_dev, mas, sizeof(double) * N * N, hipMemcpyHostToDevice);
        hipMemcpy(anew_dev, anew, sizeof(double) * N * N, hipMemcpyHostToDevice);

        while ((rep < ITER) && (err >= ACC)) //начинаем вычислять матрицу
        {
            rep++;  //итерация 1 пройдена
            countnewmatrix <<<,>>> (anew_dev, mas_dev, N); //считаю новые значения матрицы

            if (rep % 100 == 0)  //каждые 100 итераций обновляем значение ошибки
            {
                finderr <<<, >>> (mas_dev, anew_dev, errorMatrix, N * N); //ситаю значение ошибки
                hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, N * N); //ищу макс значение ошибки
                hipMemcpy(&err, deviceError, sizeof(double), hipMemcpyDeviceToHost); //обновляю значение ошибки на девайсе
            }
            cout << rep << "  " << err << endl;  //вывод итерации и значения ошибки 
        }

    cout << "Calculation time: " << 1.0 * (clock() - befca) / CLOCKS_PER_SEC << endl; //вывод времени, потраченного на вычисления матрицы
    cout << "Iteration: " << rep << " " << "Error: " << err << endl; //сколько всего итераций потребовалось и достигнутое значение ошибки
    delete[] mas;  //освобождаем память
    delete[] anew;
    hipFree(mas_dev);
    hipFree(anew_dev);
    hipFree(errorMatrix);
    hipFree(tempStorage);
    return 0; //программа завершена успешно
}
